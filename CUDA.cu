
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <string>
#include <cstring>
#include <fstream>
#include <sstream>

#define THREAD_PER_BLOCK 256
#define WORD_PER_THREAD 16

#define FIRST_PASS 0
#define SECOND_PASS 1

using namespace std;

// build the kmp table for the subsequent operations
void preKMP(char *pattern, int failure[])
{
    int m = strlen(pattern);
    int k;

    failure[0] = -1;

    for (int i = 1; i < m; i++)
    {
        k = failure[i - 1];

        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = failure[k];
        }

        failure[i] = k + 1;
    }

    return;
}

// Kernel failuretion. Implement the KMP algorithm
__global__ void KMP(char *pattern, char *target, int failure[], int answer[], int pattern_length, int target_length, int pass)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // int i = WORD_PER_THREAD * threadIdx.x;
    // int j = (WORD_PER_THREAD * (threadIdx.x + 2) - 1) >= target_length ? target_length - 1 : WORD_PER_THREAD * (threadIdx.x + 2) - 1;
    int i, j;

    switch (pass)
    {
        case FIRST_PASS:
            i = WORD_PER_THREAD * index;
            j = WORD_PER_THREAD * (index + 1) > target_length ? target_length : WORD_PER_THREAD * (index + 1);
            break;
        case SECOND_PASS:
            i = WORD_PER_THREAD * (index + 1) - (pattern_length - 1);
            j = WORD_PER_THREAD * (index + 1) + (pattern_length);
            break;
        default:
            break;
    }

    if (i >= target_length)
        return;

    __shared__ char pattern_cache[32];
    __shared__ char failure_cache[32];
    if (threadIdx.x == 0)
    {
        for (int i = 0; i < pattern_length; i++)
        {
            pattern_cache[i] = pattern[i];
            failure_cache[i] = failure[i];
        }
    }

    // __shared__ char target_cache[WORD_PER_THREAD * (THREAD_PER_BLOCK + 1)];
    // for (int i = 0; i < WORD_PER_THREAD * 2; i++)
    //     target_cache[WORD_PER_THREAD * threadIdx.x + i] = target[(WORD_PER_THREAD * index + i) >= target_length ? target_length - 1 : WORD_PER_THREAD * index + i];

    int k = 0;
    while (i < j)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern_cache[k])
        {
            i++;
            k++;

            if (k == pattern_length)
            {
                // answer[(blockIdx.x * blockDim.x * WORD_PER_THREAD) + i - pattern_length] = 1;
                k--;
                answer[i - pattern_length] = 1;
                i = i - pattern_length + 1;
            }
        }
        else
            k = failure_cache[k];
    }

    return;
}

int main(int argc, char *argv[])
{
    fstream target_file("target.txt"), pattern_file("pattern.txt");
    stringstream target_stream, pattern_stream;

    target_stream << target_file.rdbuf();
    string target_string = target_stream.str();

    pattern_stream << pattern_file.rdbuf();
    string pattern_string = pattern_stream.str();

    int target_length = target_string.length();
    int pattern_length = pattern_string.length();

    char *target = new char[target_length + 1];
    char *pattern = new char[pattern_length + 1];

    strcpy(target, target_string.c_str());
    strcpy(pattern, pattern_string.c_str());

    int *failure = new int[pattern_length];
    int *answer = new int[target_length]();

    preKMP(pattern, failure);

    char *device_target, *device_pattern;
    int *device_failure, *device_answer;

    hipEvent_t start, end;
    float elapsed_time;

    hipMalloc((void **)&device_target, target_length * sizeof(char));
    hipMalloc((void **)&device_pattern, pattern_length * sizeof(char));
    hipMalloc((void **)&device_failure, target_length * sizeof(int));
    hipMalloc((void **)&device_answer, target_length * sizeof(int));

    hipMemcpy(device_target, target, target_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(device_pattern, pattern, pattern_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(device_failure, failure, pattern_length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_answer, answer, target_length * sizeof(int), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    KMP<<<(target_length / THREAD_PER_BLOCK / WORD_PER_THREAD), THREAD_PER_BLOCK>>>(device_pattern, device_target, device_failure, device_answer, pattern_length, target_length, FIRST_PASS);
    KMP<<<(target_length / THREAD_PER_BLOCK / WORD_PER_THREAD), THREAD_PER_BLOCK>>>(device_pattern, device_target, device_failure, device_answer, pattern_length, target_length, SECOND_PASS);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    hipMemcpy(answer, device_answer, target_length * sizeof(int), hipMemcpyDeviceToHost);

    cout << "----- This is parallel results using KMP Algorithm on CUDA. -----" << endl;
    cout << "When the target length is " << target_length << ", pattern length is " << pattern_length << ", the elapsed time is " << elapsed_time << " ms." << endl;

    int counter = 0;

    for (int i = 0; i < target_length; i++)
    {
        if (answer[i])
        {
            cout << "Find a matching substring starting at: " << i << "." << endl;
            counter++;
        }
    }

    cout << counter << endl;

    hipFree(device_target);
    hipFree(device_pattern);
    hipFree(device_failure);
    hipFree(device_answer);

    delete[] target;
    delete[] pattern;
    delete[] failure;
    delete[] answer;

    return 0;
}